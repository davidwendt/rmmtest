
#include <rmm/rmm.h>
#include <memory.h>

int base()
{
   char* buffer = 0;

   hipMalloc(&buffer,100);

   const char* data = "hello, world";
   int len = (int)strlen(data)+1;
   hipMemcpy(buffer,data,len,hipMemcpyHostToDevice);

   char* output = new char[len];
   hipMemcpy(output,buffer,len,hipMemcpyDeviceToHost);
   printf("%s\n",output);

   hipFree(buffer);

   return 0;
}

int test1()
{
   char* buffer = 0;

   RMM_ALLOC(&buffer,100,0);

   const char* data = "hello, world";
   int len = (int)strlen(data)+1;
   hipMemcpy(buffer,data,len,hipMemcpyHostToDevice);

   char* output = new char[len];
   hipMemcpy(output,buffer,len,hipMemcpyDeviceToHost);
   printf("%s\n",output);

   RMM_FREE(buffer,0);

   return 0;
}


int main( int argc, char** argv )
{
   base();

   test1();

   rmmOptions_t options;
   options.allocation_mode = PoolAllocation;
   options.initial_pool_size = 0;
   options.enable_logging = false;
   rmmInitialize(&options);
   test1();
   rmmFinalize();

   options.allocation_mode = CudaManagedMemory;
   rmmInitialize(&options);
   test1();
   rmmFinalize();

   return 0;
}
