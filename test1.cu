
#include <rmm/rmm.h>
#include <memory.h>

int base()
{
   char* buffer = 0;

   hipMalloc(&buffer,100);

   const char* data = "hello, world";
   int len = (int)strlen(data)+1;
   hipMemcpy(buffer,data,len,hipMemcpyHostToDevice);

   char* output = new char[len];
   hipMemcpy(output,buffer,len,hipMemcpyDeviceToHost);
   printf("%s\n",output);

   hipFree(buffer);

   return 0;
}

int test1()
{
   char* buffer = 0;

   RMM_ALLOC(&buffer,100,0);

   const char* data = "hello, world";
   int len = (int)strlen(data)+1;
   hipMemcpy(buffer,data,len,hipMemcpyHostToDevice);

   char* output = new char[len];
   hipMemcpy(output,buffer,len,hipMemcpyDeviceToHost);
   printf("%s\n",output);

   RMM_FREE(buffer,0);

   return 0;
}

int main( int argc, char** argv )
{
   base();
   test1();
   return 0;
}
